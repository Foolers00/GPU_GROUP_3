#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////
/* TEST_PY */
/////////////////////////////////////////////////////////////////////////////////////

#ifndef TEST
#define TEST
#include "test.h"
#endif




Point_array* test_sequence_4_1(){

    Point_array* points = NULL;
    points = init_point_array(5);

    Point u, v, w, z, t;

    u.x = 0;
    u.y = 0;

    v.x = 0;
    v.y = 200;

    w.x = 100;
    w.y = 100;

    z.x = 200;
    z.y = 0;

    t.x = 200;
    t.y = 200;

    add_to_point_array(points, u);
    add_to_point_array(points, v);
    add_to_point_array(points, w);
    add_to_point_array(points, z);
    add_to_point_array(points, t);

    return points;

}


Hull* test_sequence_4_2(Point_array* points){

    return quickhull(points);

}


Point_array* test_random_generate(){
    Point_array* points = generate_random_points(15, -200, 800);
    return points;
}


Hull* test_random_hull(Point_array* points){
    return quickhull(points);
}


void initialData(size_t *init_array, const size_t size)
{

    for (size_t i = 0; i < size; ++i)
    {
        init_array[i] = rand()%100;
        //init_array[i] = size - 1 - i;
        //init_array[i] = i;
        //init_array[i] = 1;
    }
}


void compare_prescan_exclusive(size_t *h_data, size_t *gpuRef, size_t size)
{

    size_t* prefixSum = (size_t*)malloc(size*sizeof(size_t));
    if(!prefixSum){
        fprintf(stderr, "Malloc failed");
        exit(1);
    }
    bool state = true;

    prefixSum[0] = 0;
    prefixSum[1] = h_data[0];
    // Adding present element with previous element
    for (size_t i = 2; i < size; i++)
    {
        prefixSum[i] = prefixSum[i - 1] + h_data[i-1];
    }
    for (size_t i = 0; i < size; i++)
    {
        if (prefixSum[i] != gpuRef[i])
        {
            fprintf(stdout, "Prefix sum false at index %lu: expected: %lu, actual: %lu\n", i, prefixSum[i], gpuRef[i]);
            for(size_t j = i-3; j < i+3; j++){
                fprintf(stdout, "i: %lu, h_data: %lu, prefix: %lu, gpuref: %lu\n", j, h_data[j], prefixSum[j], gpuRef[j]);
            }
            state = false;
            break;
        }
    }
    if(state){
        printf("Comparison Success\n");
    }
    else{
        printf("Comparison Failed\n");
    }
}


void compare_prescan_inclusive(size_t *h_data, size_t *gpuRef, size_t size)
{
    size_t* prefixSum = (size_t*)malloc(size*sizeof(size_t));
    if(!prefixSum){
        fprintf(stderr, "Malloc failed");
        exit(1);
    }
    bool state = true;
    prefixSum[0] = h_data[0];
    // Adding present element with previous element
    for (size_t i = 1; i < size; i++)
    {
        prefixSum[i] = prefixSum[i-1] + h_data[i];
    }
    for (size_t i = 0; i < size; i++)
    {
        if (prefixSum[i] != gpuRef[i])
        {
            fprintf(stdout, "Prefix sum false at index %lu: expected: %lu, actual: %lu\n", i, prefixSum[i], gpuRef[i]);
            for(size_t j = i-3; j < i+3; j++){
                fprintf(stdout, "i: %lu, h_data: %lu, prefix: %lu, gpuref: %lu\n", j, h_data[j], prefixSum[j], gpuRef[j]);
            }
            state = false;
            break;
        }
    }
    if(state){
        printf("Comparison Success\n");
    }
    else{
        printf("Comparison Failed\n");
    }
}


void printData(size_t *data, const size_t size)
{

    fprintf(stdout, "\n");
    fprintf(stdout, "Data: ");
    for (int i = 0; i < size - 1; i++)
    {
        fprintf(stdout, "%lu, ", data[i]);
    }
    fprintf(stdout, "%lu", data[size - 1]);
    fprintf(stdout, "\n");
}


void test_prescan(){

    // clock
    clock_t tic = clock();
    clock_t toc = clock();
    double new_scan;
    double old_scan;
    double new_scan_avg;
    double old_scan_avg;
    int iterations = 100;

    size_t* array; 
    size_t array_size;
    size_t array_bytes;
    size_t* gpuRef;

    array_size = 10000000;

    array_bytes = array_size*(sizeof(size_t));

    #if MEMORY_MODEL == STD_MEMORY
        array = (size_t*)malloc(sizeof(size_t)*array_size);
        if(!array){
            fprintf(stderr, "Malloc failed");
            exit(1);
        }
    #elif MEMORY_MODEL == PINNED_MEMORY
        CHECK(hipHostMalloc((size_t**)&array, sizeof(size_t)*array_size));
    #elif MEMORY_MODEL == ZERO_MEMORY
        CHECK(hipHostAlloc((void**)&array, sizeof(size_t)*array_size, hipHostMallocMapped));
    #endif

    initialData(array, array_size);
    
    #if MEMORY_MODEL == STD_MEMORY
        gpuRef = (size_t*)malloc(array_size*sizeof(size_t));
        if(!array){
            fprintf(stderr, "Malloc failed");
            exit(1);
        }
    #else
        CHECK(hipHostMalloc((size_t**)&gpuRef, sizeof(size_t)*array_size));
    #endif
    

    memset(gpuRef, 0, array_bytes);

    for(int i = 0; i < iterations; i++){
        tic = clock();
        master_prescan(gpuRef, array, array_size, array_bytes, EXCLUSIVE);
        toc = clock();
        old_scan = (double)(toc - tic)/CLOCKS_PER_SEC;
        old_scan_avg += old_scan;

        tic = clock();
        //master_stream_prescan(gpuRef, array, array_size, array_bytes, EXCLUSIVE);
        thrust::exclusive_scan(array, array+array_size, gpuRef);
        toc = clock();
        new_scan = (double)(toc - tic)/CLOCKS_PER_SEC;
        new_scan_avg += new_scan;
    }

    old_scan_avg /= iterations;
    new_scan_avg /= iterations;

    printf("Old_scan: %f, New_scan: %f\n", old_scan, new_scan);
    //printf("Time: %f\n", new_scan);

    // compare results
    compare_prescan_exclusive(array, gpuRef, array_size);

    // free memory
    #if MEMORY_MODEL == STD_MEMORY
        free(array);
        free(gpuRef);
    #elif MEMORY_MODEL == PINNED_MEMORY || MEMORY_MODEL == ZERO_MEMORY
        CHECK(hipHostFree(array));
        CHECK(hipHostFree(gpuRef));
    #endif
   
    // reset device
    CHECK(hipDeviceReset());

}


void test_prescan_gpu(){
    
    // device set up
    int dev; 
    dev = 0;
    CHECK(hipSetDevice(dev));

    // clock
    clock_t tic = clock();
    clock_t toc = clock();
    double new_scan;
    double old_scan;
    double new_scan_avg;
    double old_scan_avg;
    int iterations = 1;

    size_t* array; 
    size_t array_size;
    size_t array_bytes;
    size_t* gpuRef;

    size_t array_grid_size;
    size_t array_rem_grid_size;
    size_t array_loop_cnt;
    size_t array_fsize;
    size_t array_fbytes;

    size_t* i_array_gpu;
    size_t* o_array_gpu;

    array_size = 1000;
    array_bytes = array_size*(sizeof(size_t));

    #if MEMORY_MODEL == STD_MEMORY
        array = (size_t*)malloc(sizeof(size_t)*array_size);
        if(!array){
            fprintf(stderr, "Malloc failed");
            exit(1);
        }
    #elif MEMORY_MODEL == PINNED_MEMORY
        CHECK(hipHostMalloc((size_t**)&array, sizeof(size_t)*array_size));
    #elif MEMORY_MODEL == ZERO_MEMORY
        CHECK(hipHostAlloc((size_t**)&array, sizeof(size_t)*array_size, hipHostMallocMapped));
    #endif

    initialData(array, array_size);

    workload_calc(&array_grid_size, &array_rem_grid_size, &array_loop_cnt,
    &array_fsize, array_size);

    array_fbytes = array_fsize*sizeof(size_t);

    #if MEMORY_MODEL == STD_MEMORY || MEMORY_MODEL == PINNED_MEMORY
        CHECK(hipMalloc((size_t **)&i_array_gpu, array_fbytes));
    #endif


    CHECK(hipMalloc((size_t **)&o_array_gpu, array_fbytes));
    
    #if MEMORY_MODEL == ZERO_MEMORY
        CHECK(hipHostGetDevicePointer((void **)&i_array_gpu, (void *)array, 0));
    #else
        CHECK(hipMemcpy(i_array_gpu, array, array_bytes, hipMemcpyHostToDevice));
    #endif

    // copy back results
    #if MEMORY_MODEL == STD_MEMORY
    gpuRef = (size_t *)malloc(array_size*sizeof(size_t));
    if(!gpuRef){
        fprintf(stderr, "Malloc failed");
        exit(1);
    }
    #else
        CHECK(hipHostMalloc((size_t**)&gpuRef, sizeof(size_t)*array_size));  
    #endif

    memset(gpuRef, 0, array_bytes);

    for(int i = 0; i < iterations; i++){
        tic = clock();
        master_prescan_gpu(o_array_gpu, i_array_gpu, array_fsize, array_fbytes, 
        array_grid_size, array_rem_grid_size, array_loop_cnt, EXCLUSIVE);
        toc = clock();
        old_scan = (double)(toc - tic)/CLOCKS_PER_SEC;
        old_scan_avg += old_scan;

        tic = clock();
        //master_stream_prescan_gpu(o_array_gpu, i_array_gpu, array_fsize, array_fbytes, 
        //array_grid_size, array_rem_grid_size, array_loop_cnt, EXCLUSIVE);
        thrust::exclusive_scan(array, array+array_size, gpuRef); //unfair
        toc = clock();
        new_scan = (double)(toc - tic)/CLOCKS_PER_SEC;
        new_scan_avg += new_scan;
    }

    old_scan_avg /= iterations;
    new_scan_avg /= iterations;

    printf("Old_scan: %f, New_scan: %f\n", old_scan, new_scan);
    //printf("Time: %f\n", new_scan);

    CHECK(hipMemcpy(gpuRef, o_array_gpu, array_bytes, hipMemcpyDeviceToHost));

    // compare results
    compare_prescan_exclusive(array, gpuRef, array_size);

    //free memory
    #if MEMORY_MODEL == STD_MEMORY || MEMORY_MODEL == PINNED_MEMORY
        CHECK(hipFree(i_array_gpu));
    #endif


    CHECK(hipFree(o_array_gpu));

    #if MEMORY_MODEL == STD_MEMORY
        free(gpuRef);
        free(array);
    #else
        CHECK(hipHostFree(gpuRef));
        CHECK(hipHostFree(array));
    #endif
    


    // reset device
    CHECK(hipDeviceReset());

}


void test_max_distance_cuda(){
    int size = 100000000;

    Point_array_par* points = init_point_array_par(size);

    Point near = (Point){.x = 1, .y = 2};
    Point far = (Point){.x = 1, .y = 8};
    Line l = (Line){.p = (Point){.x = 0, .y = 0}, .q = (Point){.x = 1000, .y = 1000}};

    for(int i = 0; i < size; i++) {
        points->array[i] = near;
        if (i == 1234) points->array[i] = far;
    }

    Line* d_l;
    Line* l_p_max, *l_max_q;

    CHECK(hipMalloc((void**)&d_l, sizeof(Line)));
    CHECK(hipMemcpy(d_l, &l, sizeof(Line), hipMemcpyHostToDevice));
    max_distance_cuda(d_l, points, &l_p_max, &l_max_q);

    Line l_p_max_host, l_max_q_host;

    CHECK(hipMemcpy(&l_p_max_host, l_p_max, sizeof(Line), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(&l_max_q_host, l_max_q, sizeof(Line), hipMemcpyDeviceToHost));
    printf("l_p_max:\tp: (%f, %f)\tq: (%f, %f)\n", l_p_max_host.p.x, l_p_max_host.p.y, l_p_max_host.q.x, l_p_max_host.q.y);
    printf("l_max_q:\tp: (%f, %f)\tq: (%f, %f)\n", l_max_q_host.p.x, l_max_q_host.p.y, l_max_q_host.q.x, l_max_q_host.q.y);
}


void test_minmax_cuda(){
    int size = 100000000;

    Point_array_par* points = init_point_array_par(size);

    Point left = (Point){.x = -1, .y = 2};
    Point middle = (Point){.x = 100, .y = 8};
    Point right = (Point){.x = 200, .y = 3};

    for(int i = 0; i < size; i++){
        if(i == 9000000){
            points->array[i] = left;
        }else if(i == 1000000){
            points->array[i] = right;
        }else{
            points->array[i] = middle;
        }
    }

    Line* minmax;
    minmax_cuda(points, &minmax);

    Line minmax_h;
    CHECK(hipMemcpy(&minmax_h, minmax, sizeof(Line), hipMemcpyDeviceToHost));

    printf("minmax:\tp: (%f, %f)\tq: (%f, %f)\n", minmax_h.p.x, minmax_h.p.y, minmax_h.q.x, minmax_h.q.y);

}


void validate_minmax(){
    int max_size = 100000000;
    int iterations = 1000;
    time_t t;
    srand((unsigned) time(&t));
    for(int i = 0; i < iterations; i++){
        int size = rand() % max_size;
        int l_bound = rand() % 1000000000;
        int u_bound = rand() % 1000000000;
        Point_array* tmp = generate_random_points(size,l_bound, u_bound);
        Point_array_par* in = init_point_array_par(size);
        in->array = tmp->array;

        Point min_seq, max_seq, min_cuda, max_cuda;

        clock_t tic = clock();
        points_on_hull(tmp, &min_seq, &max_seq);
        clock_t toc = clock();
        double time_seq = (double)(toc - tic)/CLOCKS_PER_SEC;


        Line* minmax;

        tic = clock();
        minmax_cuda(in, &minmax);
        toc = clock();
        double time_cuda = (double)(toc - tic)/CLOCKS_PER_SEC;
        Line minmax_h;
        CHECK(hipMemcpy(&minmax_h, minmax, sizeof(Line), hipMemcpyDeviceToHost));
        min_cuda.x = minmax_h.p.x;
        min_cuda.y = minmax_h.p.y;
        max_cuda.x = minmax_h.q.x;
        max_cuda.y = minmax_h.q.y;

        printf("time seq: %f, time cuda: %f\n", time_seq, time_cuda);


        bool valid = min_seq.x == min_cuda.x &&  max_seq.x == max_cuda.x;
        if(valid){
            printf("no error found so far\n size: %i, l_bound: %i, u_bound: %i. Min seq: [%f, %f], Min cuda: [%f, %f], Max seq: [%f, %f], Max cuda: [%f, %f]\n", size, l_bound, u_bound, min_seq.x, min_seq.y, min_cuda.x, min_cuda.y, max_seq.x, max_seq.y, max_cuda.x, max_cuda.y);
        }else {
            printf("found error.\n size: %i, l_bound: %i, u_bound: %i. Min seq: [%f, %f], Min cuda: [%f, %f], Max seq: [%f, %f], Max cuda: [%f, %f]\n", size, l_bound, u_bound, min_seq.x, min_seq.y, min_cuda.x, min_cuda.y, max_seq.x, max_seq.y, max_cuda.x, max_cuda.y);
            exit(1);
        }
    }
}


void test_split(){

    // device var
    int dev;

    // device set up
    dev = 0;
    CHECK(hipSetDevice(dev));

    // clock
    clock_t tic;
    clock_t toc;
    double cpu_time;
    double gpu_time;
    double thrust_time;

    // main array var
    int size;
    int l_bound;
    int u_bound;

    // point arrays cpu var
    Point_array* points_cpu;
    Point_array* points_above_cpu;
    Point_array* points_below_cpu;

    // point array gpu var
    Point_array_par* points_gpu;
    Point_array_par* points_above_gpu;
    Point_array_par* points_below_gpu;
    Point_array_par* temp_above;
    Point_array_par* temp_below;

    // thrust var
    thrust::device_vector<Point> points_thrust; 
    thrust::device_vector<Point> points_above_thrust; 
    thrust::device_vector<Point> points_below_thrust;
    thrust::device_vector<Line> l_pq_thrust;
    std::vector<Point> points_temp_above_thrust; 
    std::vector<Point> points_temp_below_thrust;

    // point on hull var
    Point p;
    Point q;
    Line l_pq;
    Line* l_pq_gpu;

    // state var for compare
    bool state = true;

    // set up array
    size = 1000000;
    l_bound = 0;
    u_bound = 100000000;

    points_cpu = generate_random_points(size, l_bound, u_bound);
    points_gpu = init_point_array_par(size);
    points_thrust.resize(size);

    // copy memory to gpu
    memcpy(points_gpu->array, points_cpu->array, sizeof(Point)*size);

    // copy memory to host thrust
    thrust::copy(&points_cpu->array[0], &points_cpu->array[size], points_thrust.begin());
     


    tic = clock();
    // init above/below arrays
    points_above_cpu = init_point_array(points_cpu->max_size/2);
    points_below_cpu = init_point_array(points_cpu->max_size/2);

    // points on hull
    //points_on_hull(points_cpu, &p, &q);
    p.x = 79.000000;
    p.y = 952.000000;
    q.x = 140.000000;
    q.y = 332.000000;
    l_pq = (Line) { .p = p, .q = q };


    // CPU Version

    for(size_t i = 0; i < points_cpu->curr_size; i++){
        int result = check_point_location(l_pq, points_cpu->array[i]);
        if(result == ON){
            continue;
        }

        if(result == ABOVE){
            add_to_point_array(points_above_cpu, points_cpu->array[i]);
        }
        else{
            add_to_point_array(points_below_cpu, points_cpu->array[i]);
        }
    }
    toc = clock();
    cpu_time = (double)(toc - tic)/CLOCKS_PER_SEC;

    // GPU Version


    points_above_gpu = init_point_array_par_gpu(0);
    points_below_gpu = init_point_array_par_gpu(0);

    CHECK(hipMalloc((Line **)&l_pq_gpu, sizeof(Line)));
    CHECK(hipMemcpy(l_pq_gpu, &l_pq, sizeof(Line), hipMemcpyHostToDevice));


    tic = clock();
    // splits array into above and below

    //split_point_array(points_gpu, points_above_gpu, points_below_gpu, l_pq_gpu);
    Point_array_par* points_gpu_gpu = init_point_array_par_gpu(points_gpu->size);
    CHECK(hipMemcpy(points_gpu_gpu->array, points_gpu->array, points_gpu->size*sizeof(Point), hipMemcpyHostToDevice));
    split_point_array_side(points_gpu_gpu, points_above_gpu, l_pq_gpu, ABOVE);
    split_point_array_side(points_gpu_gpu, points_below_gpu, l_pq_gpu, BELOW);

    toc = clock();
    gpu_time = (double)(toc - tic)/CLOCKS_PER_SEC;


    // THRUST Version
    tic = clock();

    l_pq_thrust.insert(l_pq_thrust.end(), l_pq);
    thrust_split_point_array(points_thrust, points_above_thrust, points_below_thrust, l_pq_thrust);

    toc = clock();
    thrust_time = (double)(toc - tic)/CLOCKS_PER_SEC;

    // copy back results
    temp_above = init_point_array_par(points_above_gpu->size*sizeof(Point));
    temp_below = init_point_array_par(points_below_gpu->size*sizeof(Point));

    CHECK(hipMemcpy(temp_above->array, points_above_gpu->array, points_above_gpu->size*sizeof(Point), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(temp_below->array, points_below_gpu->array, points_below_gpu->size*sizeof(Point), hipMemcpyDeviceToHost));
    
    points_temp_above_thrust.resize(points_above_thrust.size());
    points_temp_below_thrust.resize(points_below_thrust.size());
    thrust::copy(points_above_thrust.begin(), points_above_thrust.end(), points_temp_above_thrust.begin());
    thrust::copy(points_below_thrust.begin(), points_below_thrust.end(), points_temp_below_thrust.begin());


    // compare results
    printf("Above results: ");
    for(size_t i = 0; i < points_above_cpu->curr_size; i++){
        if(!compare_points(points_above_cpu->array[i], temp_above->array[i])){

            printf("x or y are not the same: x: %f, %f, y: %f, %f\n",
            points_above_cpu->array[i].x, temp_above->array[i].x,
            points_above_cpu->array[i].y, temp_above->array[i].y);

            state = false;

            break;
        }
    }

    if(state){
        printf("Comparison Success\n");
    }
    else{
       printf("Comparison Failed\n"); 
    }

    state = true;

    printf("Above results thrust: ");
    for(size_t i = 0; i < points_above_cpu->curr_size; i++){
        if(!compare_points(points_above_cpu->array[i], points_temp_above_thrust[i])){

            printf("x or y are not the same: x: %f, %f, y: %f, %f\n",
            points_above_cpu->array[i].x, points_temp_above_thrust[i].x,
            points_above_cpu->array[i].y, points_temp_above_thrust[i].y);

            state = false;

            break;
        }
    }

    if(state){
        printf("Comparison Success\n");
    }
    else{
       printf("Comparison Failed\n"); 
    }

    state = true;

    printf("Below results: ");
    for(size_t i = 0; i < points_below_cpu->curr_size; i++){
        if(!compare_points(points_below_cpu->array[i], temp_below->array[i])){
            printf("x or y are not the same: x: %f, %f, y: %f, %f\n",
            points_below_cpu->array[i].x, temp_below->array[i].x,
            points_below_cpu->array[i].y, temp_below->array[i].y);

            state = false;

            break;
        }
    }

    if(state){
        printf("Comparison Success\n");
    }
    else{
       printf("Comparison Failed\n"); 
    }

    state = true;

    printf("Below results thrust: ");
    for(size_t i = 0; i < points_below_cpu->curr_size; i++){
        if(!compare_points(points_below_cpu->array[i], points_temp_below_thrust[i])){
            printf("x or y are not the same: x: %f, %f, y: %f, %f\n",
            points_below_cpu->array[i].x, points_temp_below_thrust[i].x,
            points_below_cpu->array[i].y, points_temp_below_thrust[i].y);

            state = false;

            break;
        }
    }

    if(state){
        printf("Comparison Success\n");
    }
    else{
       printf("Comparison Failed\n"); 
    }

    state = true;

    printf("Size results: ");
    if(points_above_cpu->curr_size != points_above_gpu->size ||
        points_below_cpu->curr_size != points_below_gpu->size){
            printf("Sizes do not match: Above: %lu, %lu, Below: %lu, %lu\n",
            points_above_cpu->curr_size, points_above_gpu->size, points_below_cpu->curr_size,
            points_below_gpu->size);
            state = false;
    }

    if(state){
        printf("Comparison Success\n");
    }
    else{
       printf("Comparison Failed\n"); 
    }

    state = true;

    printf("Size results thurst: ");
    if(points_above_cpu->curr_size != points_temp_above_thrust.size() ||
        points_below_cpu->curr_size != points_temp_below_thrust.size()){
            printf("Sizes do not match: Above: %lu, %lu, Below: %lu, %lu\n",
            points_above_cpu->curr_size, points_temp_above_thrust.size(), points_below_cpu->curr_size,
            points_temp_below_thrust.size());
            state = false;
    }

    if(state){
        printf("Comparison Success\n");
    }
    else{
       printf("Comparison Failed\n"); 
    }


    if(state){
        printf("Comparison Success\n");
        printf("CPU time: %f, GPU time: %f, Thrust time: %f\n", cpu_time, gpu_time, thrust_time);
    }


    // free memory
    free(points_cpu);
    free(points_above_cpu);
    free(points_below_cpu);

    free_point_array_par(points_gpu);
    free_point_array_par_gpu(points_above_gpu);
    free_point_array_par_gpu(points_below_gpu);
    free_point_array_par(temp_above);
    free_point_array_par(temp_below);

    CHECK(hipFree(l_pq_gpu));

    // reset device
    //CHECK(hipDeviceReset());

}


void test_combinehull(){

    // device var
    int dev;

    // device set up
    dev = 0;
    CHECK(hipSetDevice(dev));

    // vars
    size_t hull_1_size;
    size_t hull_1_bytes;

    size_t hull_2_size;
    size_t hull_2_bytes;

    size_t hull_3_bytes;

    int l_bound;
    int u_bound;

    Hull* hull_1_cpu;
    Hull* hull_2_cpu;
    Hull* hull_3_cpu;

    Hull_par* hull_1_gpu;
    Hull_par* hull_2_gpu;
    Hull_par* hull_3_gpu;

    Line* temp;

    bool state = true;

    // set vars
    hull_1_size = 10;
    hull_1_bytes = sizeof(Line)*hull_1_size;
    hull_2_size = 10;
    hull_2_bytes = sizeof(Line)*hull_2_size;
    l_bound = 0;
    u_bound = rand()%100000;

    hull_1_cpu = generate_random_lines(hull_1_size, l_bound, u_bound);
    hull_2_cpu = generate_random_lines(hull_2_size, l_bound, u_bound);

    hull_1_gpu = init_hull_par_gpu(hull_1_size);
    hull_2_gpu = init_hull_par_gpu(hull_2_size);

    hull_1_gpu->size = hull_1_size;
    hull_2_gpu->size = hull_2_size;


    CHECK(hipMemcpy(hull_1_gpu->array , hull_1_cpu->array, hull_1_bytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(hull_2_gpu->array , hull_2_cpu->array, hull_2_bytes, hipMemcpyHostToDevice));

    hull_3_cpu = combine_hull(hull_1_cpu, hull_2_cpu);
    hull_3_gpu = combine_hull_par(hull_1_gpu, hull_2_gpu);

    hull_3_bytes = hull_3_gpu->size*sizeof(Line);

    temp = (Line*)malloc(hull_3_bytes);
    if(!temp){
        fprintf(stderr, "Malloc failed");
        exit(1);       
    }

    CHECK(hipMemcpy(temp , hull_3_gpu->array, hull_3_bytes, hipMemcpyDeviceToHost));

    printf("Combine result: ");
    for(int i = 0; i < hull_3_cpu->curr_size; i++){
        if(!compare_lines(hull_3_cpu->array[i], temp[i])){
            printf("Lines do not match: Point 1 cpu: (%f, %f),Point 2 cpu: (%f, %f), Point 1 gpu: (%f, %f), Point 2 gpu: (%f, %f)\n", 
            hull_3_cpu->array[i].p.x,
            hull_3_cpu->array[i].p.y,
            hull_3_cpu->array[i].q.x,
            hull_3_cpu->array[i].q.y,
            temp[i].p.x,
            temp[i].p.y,
            temp[i].q.x,
            temp[i].q.y);
            state = false;
        }
    }

    if(state){
        printf("Comparison Success\n");
    }
    else{
       printf("Comparison Failed\n"); 
    }

    printf("Size result: " );
    if(hull_3_cpu->curr_size != hull_3_gpu->size){
        printf("Sizes do not match: %lu, %lu\n", hull_3_cpu->curr_size, hull_3_gpu->size);
        state = false;
    }

    if(state){
        printf("Comparison Success\n");
    }
    else{
       printf("Comparison Failed\n"); 
    }

    // free memory
    free_hull(hull_3_cpu);
    free(temp);
    free_hull_par_gpu(hull_3_gpu);

    // reset device
    CHECK(hipDeviceReset());

}




void test_quickhull(){

    // device var
    int dev;

    // device set up
    dev = 0;
    CHECK(hipSetDevice(dev));

    // clock
    clock_t tic;
    clock_t toc;
    double cpu_time;
    double gpu_time;

    // vars
    size_t size;
    double l_bound;
    double u_bound;

    // cpu
    Point_array* points_cpu;
    Hull* hull_cpu;

    // gpu
    Point_array_par* points_gpu;
    Hull_par* hull_gpu;

    // state
    bool state = true;


    // set vars
    size = 100000000;
    l_bound = -10000000;
    u_bound = 10000000;

    
    points_cpu = init_point_array(2*size);
    points_gpu = generate_random_points_par(size, l_bound, u_bound);
    //points_gpu = generate_random_points_on_circle_par(size, 10);

    //readPointsFromCSV("points", &points_gpu);


    memcpy(points_cpu->array, points_gpu->array, points_gpu->size*sizeof(Point));
    points_cpu->curr_size = size;

    //print_point_array(points_cpu);

    //writePointArrayToCSV(points_cpu);

    tic = clock();
    hull_cpu = quickhull(points_cpu);
    toc = clock();
    cpu_time = (double)(toc - tic)/CLOCKS_PER_SEC;

    printf("CPU Finished\n");

    //writeHullArrayToCSV(hull_cpu);

    tic = clock();
    hull_gpu = quickhull_stream_par(points_gpu);
    toc = clock();
    gpu_time = (double)(toc - tic)/CLOCKS_PER_SEC;

    printf("GPU Finished\n");

    //writeHullparArrayToCSV(hull_gpu);


    bool state_2 = false;

    printf("Compare result: ");
    for(int i = 0; i < hull_cpu->curr_size; i++){
        state_2 = false;
        for(int j = 0; j < hull_gpu->size; j++){
            if(compare_lines(hull_cpu->array[i], hull_gpu->array[j])){
                state_2 = true;
            }
            else{
                if(hull_cpu->array[i].p.x == hull_cpu->array[i].q.x && hull_gpu->array[j].p.x == hull_gpu->array[j].q.x &&
                    hull_cpu->array[i].p.x == hull_gpu->array[j].p.x){
                    state_2 = true;
                }
                if(hull_cpu->array[i].p.y == hull_cpu->array[i].q.y && hull_gpu->array[j].p.y == hull_gpu->array[j].q.y &&
                    hull_cpu->array[i].p.y == hull_gpu->array[j].p.y){
                    state_2 = true;
                }
            }

        }
        if(state_2 == false){
            state = false;
            printf("This lines does not appear: Cpu: (%f, %f)-(%f, %f)\n",
                    hull_cpu->array[i].p.x, hull_cpu->array[i].p.y, hull_cpu->array[i].q.x, hull_cpu->array[i].q.y);
            for(int z = 0; z < hull_gpu->size; z++){
                printf("Gpu lines: (%f, %f)-(%f, %f)\n",  hull_gpu->array[z].p.x, hull_gpu->array[z].p.y, hull_gpu->array[z].q.x, hull_gpu->array[z].q.y);
            }
        }
        
    }


    if(state){
        printf("Comparison Success\n");
    }
    else{
       printf("Comparison Failed\n"); 
    }


    printf("Size result: ");
    if(points_cpu->curr_size != points_gpu->size){
        printf("Sizes do not match: CPU: %lu, GPU: %lu\n", points_cpu->curr_size, points_gpu->size);
        state = false;
    }

    if(state){
        printf("Comparison Success\n");
    }
    else{
       printf("Comparison Failed\n"); 
    }


    if(state){
        printf("Comparison Success\n");
        printf("CPU time: %f, GPU time: %f\n", cpu_time, gpu_time);
    }
   

    // free memory
    free_point_array(points_cpu);
    free_point_array_par(points_gpu);
    free_hull(hull_cpu);
    free_hull_par(hull_gpu);


    //}

    // reset device
    CHECK(hipDeviceReset());
}







void test_thrust_quickhull(){

    // device var
    int dev;

    // device set up
    dev = 0;
    CHECK(hipSetDevice(dev));

    // clock
    clock_t tic;
    clock_t toc;
    double cpu_time;
    double gpu_time;

    // vars
    size_t size;
    double l_bound;
    double u_bound;

    // cpu
    Point_array* points_cpu;
    Hull* hull_cpu;

    // gpu
    Point_array_par* points_gpu;
    Hull_par* hull_gpu;

    // state
    bool state = true;


    // set vars
    size = 10000;
    l_bound = 0;
    u_bound = 1000;

    points_cpu = init_point_array(2*size);
    points_gpu = generate_random_points_par(size, l_bound, u_bound);

    //readPointsFromCSV("points", &points_gpu);


    memcpy(points_cpu->array, points_gpu->array, points_gpu->size*sizeof(Point));
    points_cpu->curr_size = size;

    //writePointArrayToCSV(points_cpu);

    tic = clock();
    hull_cpu = quickhull(points_cpu);
    toc = clock();
    cpu_time = (double)(toc - tic)/CLOCKS_PER_SEC;

    //writeHullArrayToCSV(hull_cpu);

    thrust::host_vector<Line> hull_gpu_vec;
    tic = clock();
    thrust_quickhull(points_gpu, hull_gpu_vec);
    toc = clock();
    gpu_time = (double)(toc - tic)/CLOCKS_PER_SEC;

    hull_gpu = init_hull_par(hull_gpu_vec.size());
    for (int i = 0; i < hull_gpu_vec.size(); i++){
        hull_gpu->array[i] = hull_gpu_vec[i];
    }

    //writeHullparArrayToCSV(hull_gpu);

    bool state_2 = false;

    printf("Compare result: ");
    for(int i = 0; i < hull_cpu->curr_size; i++){
        state_2 = false;
        for(int j = 0; j < hull_gpu->size; j++){
            if(compare_lines(hull_cpu->array[i], hull_gpu->array[j])){
                state_2 = true;
            }
            else{
                if(hull_cpu->array[i].p.x == hull_cpu->array[i].q.x && hull_gpu->array[j].p.x == hull_gpu->array[j].q.x &&
                   hull_cpu->array[i].p.x == hull_gpu->array[j].p.x){
                    state_2 = true;
                }
                if(hull_cpu->array[i].p.y == hull_cpu->array[i].q.y && hull_gpu->array[j].p.y == hull_gpu->array[j].q.y &&
                   hull_cpu->array[i].p.y == hull_gpu->array[j].p.y){
                    state_2 = true;
                }
            }

        }
        if(state_2 == false){
            state = false;
            printf("This lines does not appear: Cpu: (%f, %f)-(%f, %f)\n",
                   hull_cpu->array[i].p.x, hull_cpu->array[i].p.y, hull_cpu->array[i].q.x, hull_cpu->array[i].q.y);
            for(int z = 0; z < hull_gpu->size; z++){
                printf("Gpu lines: (%f, %f)-(%f, %f)\n",  hull_gpu->array[z].p.x, hull_gpu->array[z].p.y, hull_gpu->array[z].q.x, hull_gpu->array[z].q.y);
            }
        }

    }


    if(state){
        printf("Comparison Success\n");
    }
    else{
        printf("Comparison Failed\n");
    }


    printf("Size result: ");
    if(points_cpu->curr_size != points_gpu->size){
        printf("Sizes do not match: CPU: %lu, GPU: %lu\n", points_cpu->curr_size, points_gpu->size);
        state = false;
    }

    if(state){
        printf("Comparison Success\n");
    }
    else{
        printf("Comparison Failed\n");
    }


    if(state){
        printf("Comparison Success\n");
        printf("CPU time: %f, GPU time: %f\n", cpu_time, gpu_time);
    }


    // free memory
    free_point_array(points_cpu);
    free_point_array_par(points_gpu);
    free_hull(hull_cpu);
    free_hull_par(hull_gpu);

    //}

    // reset device
    CHECK(hipDeviceReset());
}




void test_quickhull_performance(){

    // device var
    int dev;

    // device set up
    dev = 0;
    CHECK(hipSetDevice(dev));

    // clock
    clock_t tic;
    clock_t toc;
    double cpu_time;
    double gpu_time;
    double thrust_gpu_time;
    double cpu_time_avg;
    double gpu_time_avg;
    double thrust_gpu_time_avg;
    int iterations;

    // vars
    size_t size;
    double l_bound;
    double u_bound;

    // cpu
    Point_array* points_cpu;
    Hull* hull_cpu;

    // gpu / thrust
    Point_array_par* points_gpu;
    Hull_par* hull_gpu;


    // set vars
    size = 100000;
    l_bound = INT_MIN;
    u_bound = INT_MAX;

    cpu_time_avg = 0;
    gpu_time_avg = 0;
    thrust_gpu_time_avg = 0;
    iterations = 0;

    while(iterations < 3){

        points_cpu = init_point_array(2*size);
        //points_gpu = generate_random_points_par(size, l_bound, u_bound);
        points_gpu = generate_random_points_on_circle_par(size, 1000);


        memcpy(points_cpu->array, points_gpu->array, points_gpu->size*sizeof(Point));
        points_cpu->curr_size = size;

        // cpu
        tic = clock();
        hull_cpu = quickhull(points_cpu);
        toc = clock();
        cpu_time = (double)(toc - tic)/CLOCKS_PER_SEC;

        // gpu
        tic = clock();
        hull_gpu = quickhull_par(points_gpu);
        toc = clock();
        gpu_time = (double)(toc - tic)/CLOCKS_PER_SEC;

        // thrust
        thrust::host_vector<Line> hull_gpu_vec;
        tic = clock();
        thrust_quickhull(points_gpu, hull_gpu_vec);
        toc = clock();
        thrust_gpu_time = (double)(toc - tic)/CLOCKS_PER_SEC;

        printf("CPU Size: %lu, GPU Size: %lu, Thrust Size: %lu\n", hull_cpu->curr_size, hull_gpu->size, hull_gpu_vec.size());


        // free memory
        free_point_array(points_cpu);
        free_point_array_par(points_gpu);
        free_hull(hull_cpu);
        free_hull_par(hull_gpu);

        cpu_time_avg += cpu_time;
        gpu_time_avg += gpu_time;
        thrust_gpu_time_avg += thrust_gpu_time;
        iterations++;

    }

    cpu_time_avg/=iterations;
    gpu_time_avg/=iterations;
    thrust_gpu_time_avg/=iterations;
    
    printf("CPU time: %f, GPU time: %f, GPU Thrust time: %f\n", cpu_time_avg, 
    gpu_time_avg, thrust_gpu_time_avg);
    
    // reset device
    CHECK(hipDeviceReset());
}

void test_quickhull_performance(size_t size, FILE* output_file){
    // device var
    int dev;

    // device set up
    dev = 0;
    CHECK(hipSetDevice(dev));

    // clock
    clock_t tic;
    clock_t toc;
    double cpu_time;
    double gpu_time;
    double thrust_gpu_time;
    double cpu_time_avg;
    double gpu_time_avg;
    double thrust_gpu_time_avg;
    int iterations;

    // vars
    double l_bound;
    double u_bound;

    // cpu
    Point_array* points_cpu;
    Hull* hull_cpu;

    // gpu / thrust
    Point_array_par* points_gpu;
    Hull_par* hull_gpu;


    // set vars
    l_bound = -1;
    u_bound = 1;

    cpu_time_avg = 0;
    gpu_time_avg = 0;
    thrust_gpu_time_avg = 0;
    iterations = 0;

    while(iterations < 100){

        points_cpu = init_point_array(2*size);
        points_gpu = generate_random_points_par(size, l_bound, u_bound);


        memcpy(points_cpu->array, points_gpu->array, points_gpu->size*sizeof(Point));
        points_cpu->curr_size = size;

        // cpu
        tic = clock();
        hull_cpu = quickhull(points_cpu);
        toc = clock();
        cpu_time = (double)(toc - tic)/CLOCKS_PER_SEC;

        // gpu
        tic = clock();
        hull_gpu = quickhull_par(points_gpu);
        toc = clock();
        gpu_time = (double)(toc - tic)/CLOCKS_PER_SEC;

        // thrust
        thrust::host_vector<Line> hull_gpu_vec;
        tic = clock();
        thrust_quickhull(points_gpu, hull_gpu_vec);
        toc = clock();
        thrust_gpu_time = (double)(toc - tic)/CLOCKS_PER_SEC;


        // free memory
        free_point_array(points_cpu);
        free_point_array_par(points_gpu);
        free_hull(hull_cpu);
        free_hull_par(hull_gpu);

        cpu_time_avg += cpu_time;
        gpu_time_avg += gpu_time;
        thrust_gpu_time_avg += thrust_gpu_time;
        iterations++;

    }

    cpu_time_avg/=iterations;
    gpu_time_avg/=iterations;
    thrust_gpu_time_avg/=iterations;
    
    // printf("CPU time: %f, GPU time: %f, GPU Thrust time: %f\n", cpu_time_avg, 
    // gpu_time_avg, thrust_gpu_time_avg);
    fprintf(output_file, "CPU time: %f, GPU time: %f, GPU Thrust time: %f, MEMORY_MODEL: %d, BLOCKSIZE: %d, size: %zu\n",
        cpu_time_avg, gpu_time_avg, thrust_gpu_time_avg, MEMORY_MODEL, BLOCKSIZE, size);



    // reset device
    CHECK(hipDeviceReset());
}



void writePointArrayToCSV(Point_array* points){
    FILE *file = fopen("points", "w");
    if (file == NULL) {
        printf("Error opening file: %s\n", "points");
        return;
    }

    for (int i = 0; i < points->curr_size; i++) {
        fprintf(file, "%f,%f\n", points->array[i].x,points->array[i].y);
    }

    fclose(file);
    printf("Array successfully written to CSV file: %s\n", "points");
}


void writeHullArrayToCSV(Hull* hull) {
    FILE *file = fopen("cpu_hull", "w");
    if (file == NULL) {
        printf("Error opening file: %s\n", "cpu_hull");
        return;
    }

    for (int i = 0; i < hull->curr_size; i++) {
        fprintf(file, "%f,%f,%f,%f\n", hull->array[i].p.x,hull->array[i].p.y,hull->array[i].q.x,hull->array[i].q.y);
    }

    fclose(file);
    printf("Array successfully written to CSV file: %s\n", "cpu_hull");
}


void writeHullparArrayToCSV(Hull_par* hull) {
    FILE *file = fopen("gpu_hull", "w");
    if (file == NULL) {
        printf("Error opening file: %s\n", "gpu_hull");
        return;
    }

    for (int i = 0; i < hull->size; i++) {
        fprintf(file, "%f,%f,%f,%f\n", hull->array[i].p.x,hull->array[i].p.y,hull->array[i].q.x,hull->array[i].q.y);
    }

    fclose(file);
    printf("Array successfully written to CSV file: %s\n", "gpu_hull");
}





void readPointsFromCSV(const char* filename, Point_array_par** points) {
    FILE* file = fopen(filename, "r");
    if (file == NULL) {
        printf("Error opening file: %s\n", filename);
    }

    // Count the number of lines in the file
    int lines = 0;
    char ch;
    while (!feof(file)) {
        ch = fgetc(file);
        if (ch == '\n') {
            lines++;
        }
    }
    rewind(file);  // Reset the file pointer to the beginning

    // Allocate memory for the points
    *points = (Point_array_par*)malloc(sizeof(Point_array_par));
    if(!points){
        printf("Malloc failed");
    }

    (*points)->array = (Point*)malloc(lines * sizeof(Point));
    if(!(*points)->array){
        printf("Malloc failed");
    }

    // Read points from the file
    int i = 0;
    while (fscanf(file, "%lf,%lf", &(*points)->array[i].x, &(*points)->array[i].y) != EOF) {
        i++;
    }

    fclose(file);
    (*points)->size = lines;
    printf("Points successfully read from CSV file: %s\n", filename);
}




Hull* generate_random_lines(int num_of_lines, double l_bound, double u_bound){

    time_t t;
    double difference = u_bound - l_bound;
    double offset_x_1 = 0;
    double offset_y_1 = 0;
    double offset_x_2 = 0;
    double offset_y_2 = 0;
    Point point_1;
    Point point_2;
    Line l;
    srand((unsigned) time(&t));

    Hull* hull = init_hull(num_of_lines * 2);
    for(size_t i = 0; i < num_of_lines; i++){
        offset_x_1 = rand() % (int)difference;
        offset_y_1 = rand() % (int)difference;
        offset_x_2 = rand() % (int)difference;
        offset_y_2 = rand() % (int)difference;
        point_1 = (Point) {.x = l_bound + offset_x_1, .y = l_bound + offset_y_1};
        point_2 = (Point) {.x = l_bound + offset_x_2, .y = l_bound + offset_y_2};
        l = init_line(point_1, point_2);
        add_to_hull(hull, l);
    }

    return hull;
}



void test_memory_model() {

    // // Set up different memory models to test
    // int memory_models[] = {STD_MEMORY, PINNED_MEMORY, ZERO_MEMORY};
    // int num_models = sizeof(memory_models) / sizeof(memory_models[0]);

    // Print the value of BLOCKSIZE
    std::cout << "BLOCKSIZE: " << BLOCKSIZE << std::endl;

    // Define the maximum input size
    int max_input_size = 1000000;

    std::string fileSuffix = "_" + std::to_string(MEMORY_MODEL) + "_" + std::to_string(BLOCKSIZE);
    std::string fileName = "test_memory_output" + fileSuffix + ".txt";
    
    FILE* output_file = fopen(fileName.c_str(), "w");    
    if (output_file == NULL) {
        printf("Error opening output file.\n");
        return;
    }

    // for (int i = 0; i < num_models; i++) {
    //     // Set MEMORY_MODEL to current memory model
    //     #undef MEMORY_MODEL
    //     #define MEMORY_MODEL memory_models[i]

    //     // Perform testing with current memory model
    //     printf("Testing with MEMORY_MODEL: %d\n", MEMORY_MODEL);

    for (int size = 10; size <= max_input_size; size *= 10) {
        printf("Testing with input size: %d\n", size);

        // Call the function and capture the output
        test_quickhull_performance(size, output_file);

        // fprintf(output_file, "CPU time: %f, GPU time: %f, GPU Thrust time: %f, MEMORY_MODEL: %d, BLOCKSIZE: %d, size: %zu\n",
        //         cpu_time_avg, gpu_time_avg, thrust_gpu_time_avg, MEMORY_MODEL, BLOCKSIZE, size);
    
    }

    fclose(output_file);  // Close the output file
}
