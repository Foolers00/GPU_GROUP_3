#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////
/* MAX DISTANCE CUDA */
/////////////////////////////////////////////////////////////////////////////////////

#ifndef MINMAX
#define MINMAX
#include "minmax.h"
#endif

#ifndef TEST
#define TEST
#include "../Test/test.h"
#endif


__global__ void minmax_kernel(minmaxPoint points, int size, minmaxPoint result){

    // 1024 * 16 * 2 = 32,8 KB
    __shared__ Point sdata_min[1024];
    __shared__ Point sdata_max[1024];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if(gid < size){
        sdata_min[tid] = points.min[gid];
        sdata_max[tid] = points.max[gid];
    }else{
        sdata_min[tid] = points.min[0]; // dummy value
        sdata_max[tid] = points.max[0]; // dummy value
    }

    __syncthreads();

    for(int s = blockDim.x / 2; s > 0; s >>= 1){
        if(tid < s){
            //find Max
            if(sdata_max[tid].x < sdata_max[tid + s].x){
                sdata_max[tid] = sdata_max[tid + s];
            }
            //find Min
            if(sdata_min[tid].x > sdata_min[tid + s].x) {
                sdata_min[tid] = sdata_min[tid + s];
            }
        }
        __syncthreads();
    }

    if(tid == 0){
        result.min[blockIdx.x] = sdata_min[0];
        result.max[blockIdx.x] = sdata_max[0];
    }
}

//TODO: calculate both min and max, not only max.
void minmax_cuda(Point_array* points, Point* min, Point* max){
    int size = points->curr_size;
    int threadsPerBlock = 1024; //!!! always power of two and max 1024 because of static size of shared array in kernel !!!
    int numBlocks = (size + threadsPerBlock - 1)/threadsPerBlock;

    minmaxPoint points_in;
    CHECK(hipMalloc((void**)&(points_in.max), size * sizeof(Point)));
    CHECK(hipMalloc((void**)&(points_in.min), size * sizeof(Point)));
    CHECK(hipMemcpy(points_in.max, points->array, size*sizeof(Point), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(points_in.min, points->array, size*sizeof(Point), hipMemcpyHostToDevice));


    minmaxPoint points_out;
    CHECK(hipMalloc((void**)&(points_out.max), numBlocks * sizeof(Point)));
    CHECK(hipMalloc((void**)&(points_out.min), numBlocks * sizeof(Point)));

    while(size > threadsPerBlock){

        minmax_kernel<<<numBlocks, threadsPerBlock>>>(points_in, size, points_out);
        CHECK(hipMemcpy(points_in.max, points_out.max, numBlocks * sizeof(Point), hipMemcpyDeviceToDevice));
        CHECK(hipMemcpy(points_in.min, points_out.min, numBlocks * sizeof(Point), hipMemcpyDeviceToDevice));
        size = numBlocks;
        numBlocks = (size + threadsPerBlock - 1)/threadsPerBlock;
//        CHECK(hipFree(d_points_out->min));
//        CHECK(hipFree(d_points_out->max));
//        CHECK(hipFree(d_points_out));
//        CHECK(hipMalloc((void**)&d_points_out_min, numBlocks * sizeof(Point)));
//        CHECK(hipMalloc((void**)&d_points_out_max, numBlocks * sizeof(Point)));
    }

    //deal with the rest
    minmax_kernel<<<1, threadsPerBlock>>>(points_in, size, points_out);

    CHECK(hipMemcpy(max, points_out.max, sizeof(Point), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(min, points_out.min, sizeof(Point), hipMemcpyDeviceToHost));

    CHECK(hipFree(points_out.min));
    CHECK(hipFree(points_out.max));
    CHECK(hipFree(points_in.min));
    CHECK(hipFree(points_in.max));

}
//int main(int argc, char **argv){
//    int size = 450000000; // 450 mio
//
//    Point_array* points = init_point_array(size);
//
//    Point left = (Point){.x = -1, .y = 2};
//    Point middle = (Point){.x = 100, .y = 8};
//    Point right = (Point){.x = 200, .y = 3};
//
//    for(int i = 0; i < size; i++){
//        if(i == 90000000){ // 90 Mio
//            add_to_point_array(points, left);
//        }else if(i == 1000000){ // 1 Mio
//            add_to_point_array(points, right);
//        }else{
//            add_to_point_array(points, middle);
//        }
//    }
//
//    time_t tic = clock();
//    Point max_cuda, min_cuda;
//    minmax_cuda(points, &min_cuda, &max_cuda);
//    time_t toc = clock();
//    double sec_cuda = (double)(toc - tic)/CLOCKS_PER_SEC;
//    printf("Max cuda: (%f, %f), Min cuda: (%f, %f), Time elapsed: %f\n", max_cuda.x, max_cuda.y, min_cuda.x, min_cuda.y, sec_cuda);
//
//    tic = clock();
//    Point max_seq, min_seq;
//    points_on_hull(points, &min_seq, &max_seq);
//    toc = clock();
//    double sec_seq = (double)(toc - tic)/CLOCKS_PER_SEC;
//    printf("Max seq: (%f, %f), Min seq: (%f, %f), Time elapsed: %f\n", max_seq.x, max_seq.y, min_seq.x, min_seq.y, sec_seq);
//
//    return 0;
//}
