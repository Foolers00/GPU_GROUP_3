#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////
/* MAX DISTANCE CUDA */
/////////////////////////////////////////////////////////////////////////////////////

#ifndef MINMAX
#define MINMAX
#include "minmax.h"
#endif

#ifndef TEST
#define TEST
#include "../Test/test.h"
#endif





__global__ void minmax_kernel(minmaxPoint points, int size, minmaxPoint result){

    // 1024 * 16 * 2 = 32,8 KB
    __shared__ Point sdata_min[BLOCKSIZE];
    __shared__ Point sdata_max[BLOCKSIZE];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if(gid < size){
        sdata_min[tid] = points.min[gid];
        sdata_max[tid] = points.max[gid];
    }else{
        sdata_min[tid] = points.min[0]; // dummy value
        sdata_max[tid] = points.max[0]; // dummy value
    }

    __syncthreads();

    for(int s = blockDim.x / 2; s > 0; s >>= 1){
        if(tid < s){
            //find Max
            if(sdata_max[tid].x < sdata_max[tid + s].x){
                sdata_max[tid] = sdata_max[tid + s];
            }
            //find Min
            if(sdata_min[tid].x > sdata_min[tid + s].x) {
                sdata_min[tid] = sdata_min[tid + s];
            }
        }
        __syncthreads();
    }

    if(tid == 0){
        result.min[blockIdx.x] = sdata_min[0];
        result.max[blockIdx.x] = sdata_max[0];
    }
}


__global__ void assign_max_lines_par(Line* l_ptr, Point* min_ptr, Point* max_ptr){

    if(threadIdx.x == 0){
        l_ptr->p = *min_ptr;
        l_ptr->q = *max_ptr;
    }
}

void minmax_cuda(Point_array_par* points, Line** l_pq){
    int size = points->size;
    int threadsPerBlock = BLOCKSIZE; //!!! always power of two and max 1024 because of static size of shared array in kernel !!!
    int numBlocks = (size + threadsPerBlock - 1)/threadsPerBlock;

    minmaxPoint points_in;
    CHECK(hipMalloc((void**)&(points_in.max), size * sizeof(Point)));
    CHECK(hipMalloc((void**)&(points_in.min), size * sizeof(Point)));
    CHECK(hipMemcpy(points_in.max, points->array, size*sizeof(Point), hipMemcpyDeviceToDevice));
    CHECK(hipMemcpy(points_in.min, points->array, size*sizeof(Point), hipMemcpyDeviceToDevice));


    minmaxPoint points_out;
    CHECK(hipMalloc((void**)&(points_out.max), numBlocks * sizeof(Point)));
    CHECK(hipMalloc((void**)&(points_out.min), numBlocks * sizeof(Point)));

    while(size > threadsPerBlock){

        minmax_kernel<<<numBlocks, threadsPerBlock>>>(points_in, size, points_out);
        CHECK(hipMemcpy(points_in.max, points_out.max, numBlocks * sizeof(Point), hipMemcpyDeviceToDevice));
        CHECK(hipMemcpy(points_in.min, points_out.min, numBlocks * sizeof(Point), hipMemcpyDeviceToDevice));
        size = numBlocks;
        numBlocks = (size + threadsPerBlock - 1)/threadsPerBlock;
    }

    //deal with the rest
    minmax_kernel<<<1, threadsPerBlock>>>(points_in, size, points_out);

    // allocate GPU mem at address handed over as arguments
    CHECK(hipMalloc(l_pq, sizeof(Line)));
    assign_max_lines_par<<<1, 1>>>(*l_pq, points_out.min, points_out.max);

    CHECK(hipFree(points_out.min));
    CHECK(hipFree(points_out.max));
    CHECK(hipFree(points_in.min));
    CHECK(hipFree(points_in.max));

}


///////////////////////////////////////////////////////////////////////////////
// Stream functions


void minmax_stream_cuda(Point_array_par* points, Line** l_pq, hipStream_t* streams){
    int size = points->size;
    int threadsPerBlock = BLOCKSIZE; //!!! always power of two and max 1024 because of static size of shared array in kernel !!!
    int numBlocks = (size + threadsPerBlock - 1)/threadsPerBlock;

    minmaxPoint points_in;
    CHECK(hipMallocAsync((void**)&(points_in.max), size * sizeof(Point), streams[0]));
    CHECK(hipMallocAsync((void**)&(points_in.min), size * sizeof(Point), streams[1]));

    // Synchronice Kernels
    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);

    CHECK(hipMemcpyAsync(points_in.max, points->array, size*sizeof(Point), hipMemcpyDeviceToDevice, streams[0]));
    CHECK(hipMemcpyAsync(points_in.min, points->array, size*sizeof(Point), hipMemcpyDeviceToDevice, streams[1]));


    minmaxPoint points_out;
    CHECK(hipMallocAsync((void**)&(points_out.max), numBlocks * sizeof(Point), streams[0]));
    CHECK(hipMallocAsync((void**)&(points_out.min), numBlocks * sizeof(Point), streams[1]));

    // Synchronice Kernels
    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);

    while(size > threadsPerBlock){

        minmax_kernel<<<numBlocks, threadsPerBlock, 0, streams[0]>>>(points_in, size, points_out);

        // Synchronice Kernels
        hipStreamSynchronize(streams[0]);

        CHECK(hipMemcpyAsync(points_in.max, points_out.max, numBlocks * sizeof(Point), hipMemcpyDeviceToDevice, streams[0]));
        CHECK(hipMemcpyAsync(points_in.min, points_out.min, numBlocks * sizeof(Point), hipMemcpyDeviceToDevice, streams[1]));
        size = numBlocks;
        numBlocks = (size + threadsPerBlock - 1)/threadsPerBlock;
        
        // Synchronice Kernels
        hipStreamSynchronize(streams[0]);
        hipStreamSynchronize(streams[1]);
    }

    //deal with the rest
    minmax_kernel<<<1, threadsPerBlock, 0, streams[0]>>>(points_in, size, points_out);

    // allocate GPU mem at address handed over as arguments
    CHECK(hipMallocAsync(l_pq, sizeof(Line), streams[0]));
    assign_max_lines_par<<<1, 1, 0, streams[0]>>>(*l_pq, points_out.min, points_out.max);

    // Synchronice Kernels
    hipStreamSynchronize(streams[0]);

    // free memory
    CHECK(hipFreeAsync(points_in.min, streams[0]));
    CHECK(hipFreeAsync(points_in.max, streams[1]));
    CHECK(hipFreeAsync(points_out.min, streams[0]));
    CHECK(hipFreeAsync(points_out.max, streams[1]));

    // Synchronice Kernels
    hipStreamSynchronize(streams[0]);
    hipStreamSynchronize(streams[1]);


}

///////////////////////////////////////////////////////////////////////////////


//int main(int argc, char** argv){
//
//    int size = 100000000;
//
//    Point_array_par* points = init_point_array_par(size);
//
//    Point left = (Point){.x = -1, .y = 2};
//    Point middle = (Point){.x = 100, .y = 8};
//    Point right = (Point){.x = 200, .y = 3};
//
//    for(int i = 0; i < size; i++){
//        if(i == 9000000){
//            points->array[i] = left;
//        }else if(i == 1000000){
//            points->array[i] = right;
//        }else{
//            points->array[i] = middle;
//        }
//    }
//
//    Line* minmax;
//    minmax_cuda(points, &minmax);
//
//    Line minmax_h;
//    CHECK(hipMemcpy(&minmax_h, minmax, sizeof(Line), hipMemcpyDeviceToHost));
//
//    printf("minmax:\tp: (%f, %f)\tq: (%f, %f)\n", minmax_h.p.x, minmax_h.p.y, minmax_h.q.x, minmax_h.q.y);
//
//    return 0;
//}