#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////
/* GENERAL FUNCTIONS PARALLEL */
/////////////////////////////////////////////////////////////////////////////////////


#ifndef GENERAL_FUNCTIONS_PAR
#define GENERAL_FUNCTIONS_PAR
#include "general_functions_par.h"
#endif

#ifndef SPLIT
#define SPLIT
#include "split.h"
#endif



Hull_par* quickhull_par(Point_array_par* points){

    // device var
    int dev;

    // device set up
    dev = 0;
    CHECK(hipSetDevice(dev));

    Line* l_pq = NULL; //device pointer
    Hull_par* hull_up = NULL;
    Hull_par* hull_down = NULL;
    Hull_par* hull_result_gpu = NULL;
    Hull_par* hull_result_cpu = NULL;

    // above below array var
    Point_array_par* points_gpu;
    Point_array_par* points_above;
    Point_array_par* points_below;


    // set memory
    points_above = init_point_array_par_gpu(0);
    points_below = init_point_array_par_gpu(0);

    // transfer points
    #if MEMORY_MODEL == ZERO_MEMORY
        points_gpu = init_point_array_par_gpu(0);
        points_gpu->size = points->size;
        CHECK(hipHostGetDevicePointer((void **)&points_gpu->array, (void *)points->array, 0));
    #else
        points_gpu = init_point_array_par_gpu(points->size);
        CHECK(hipMemcpy(points_gpu->array, points->array, points->size*sizeof(Point), hipMemcpyHostToDevice));
    #endif



    // find points on hull
    points_on_hull_par(points_gpu, &l_pq);
    //l_pq = (Line) { .p = p, .q = q };

    // Line* l_pq_h = (Line*)malloc(sizeof(Line));
    // CHECK(hipMemcpy(l_pq_h, l_pq, sizeof(Line), hipMemcpyDeviceToHost));
    // printf("Line: (%f-%f)-(%f-%f)\n", l_pq_h->p.x, l_pq_h->p.y, l_pq_h->q.x, l_pq_h->q.y);


    // splits array into above and below
    split_point_array(points_gpu, points_above, points_below, l_pq);

    // free memory
    #if MEMORY_MODEL != ZERO_MEMORY
        free_point_array_par_gpu(points_gpu);
    #endif


    // Point_array_par points_above_h, points_below_h;
    // points_above_h.array = (Point*)malloc(points_above->size*sizeof(Point));
    // points_below_h.array = (Point*)malloc(points_below->size*sizeof(Point));
    // points_above_h.size = points_above->size;
    // points_below_h.size = points_below->size;

    // printf("size above: %lu\n", points_above->size);
    // printf("size below: %lu\n", points_below->size); 

    // printf("size above: %lu\n", points_above_h.size);
    // printf("size below: %lu\n", points_below_h.size); 

    // CHECK(hipMemcpy(points_above_h.array, points_above->array, points_above->size*sizeof(Point), hipMemcpyDeviceToHost));
    // CHECK(hipMemcpy(points_below_h.array, points_below->array, points_below->size*sizeof(Point), hipMemcpyDeviceToHost));

    // printf("size above: %lu\n", points_above->size);
    // printf("size below: %lu\n", points_below->size); 

    // printf("size above: %lu\n", points_above_h.size);
    // printf("size below: %lu\n", points_below_h.size); 
    
    // print_point_array_par(&points_above_h);
    // print_point_array_par(&points_below_h);

    
    

    // recursive call
    hull_up = first_quickhull_split_par(points_above, l_pq, ABOVE);
    hull_down = first_quickhull_split_par(points_below, l_pq, BELOW);

    // combine
    hull_result_gpu = combine_hull_par(hull_up, hull_down);

    // copy back results
    hull_result_cpu = init_hull_par(hull_result_gpu->size);
    CHECK(hipMemcpy(hull_result_cpu->array, hull_result_gpu->array, hull_result_gpu->size*sizeof(Line), hipMemcpyDeviceToHost));
    
    // free memory
    free_line_par_gpu(l_pq);
    free_hull_par_gpu(hull_result_gpu);
    free_point_array_par_gpu(points_above);
    free_point_array_par_gpu(points_below);

    return hull_result_cpu;

}

Hull_par* first_quickhull_split_par(Point_array_par* points, Line* l, int side){


    // vars
    Point_array_par* points_side = NULL;
    Line* l_p_max = NULL;
    Line* l_max_q = NULL;
    Hull_par* hull_side = NULL;

    // set memory
    points_side = points;

    // find point with max distance
    max_distance_cuda(l, points_side, &l_p_max, &l_max_q); // returns l_p_max and l_max_q gpu mem pointer, l is a device pointer
    // l_p_max = (Line) { .p = l.p, .q = max_point };
    // l_max_q = (Line) { .p = max_point, .q = l.q };


    if(points_side->size == 0) {
        hull_side = init_hull_par_gpu(1);
        CHECK(hipMemcpy(hull_side->array , l, sizeof(Line), hipMemcpyDeviceToDevice));
        hull_side->size = 1;
    }else if(points_side->size == 1){
        hull_side = init_hull_par_gpu(2);
        CHECK(hipMemcpy(hull_side->array , l_p_max, sizeof(Line), hipMemcpyDeviceToDevice));
        CHECK(hipMemcpy(hull_side->array+1 , l_max_q, sizeof(Line), hipMemcpyDeviceToDevice));
        hull_side->size = 2;

    }else {
        //points_side->curr_size > 1
        hull_side = combine_hull_par(
                quickhull_split_par(points_side, l_p_max, side),
                quickhull_split_par(points_side, l_max_q, side)
        );
    }

    // free memory
    free_line_par_gpu(l_p_max);
    free_line_par_gpu(l_max_q);

    return hull_side;

}


Hull_par* quickhull_split_par(Point_array_par* points, Line* l, int side){

    // vars
    Point_array_par* points_side = NULL;
    Line* l_p_max = NULL;
    Line* l_max_q = NULL;
    Hull_par* hull_side = NULL;

    // set memory
    points_side = init_point_array_par_gpu(0);

    // split array
    split_point_array_side(points, points_side, l, side);

    // find point with max distance
    max_distance_cuda(l, points_side, &l_p_max, &l_max_q); // returns l_p_max and l_max_q gpu mem pointer
    // l_p_max = (Line) { .p = l.p, .q = max_point };
    // l_max_q = (Line) { .p = max_point, .q = l.q };

    if(points_side->size == 0) {
        hull_side = init_hull_par_gpu(1);
        CHECK(hipMemcpy(hull_side->array , l, sizeof(Line), hipMemcpyDeviceToDevice));
        hull_side->size = 1;
    }else if(points_side->size == 1){
        hull_side = init_hull_par_gpu(2);
        CHECK(hipMemcpy(hull_side->array , l_p_max, sizeof(Line), hipMemcpyDeviceToDevice));
        CHECK(hipMemcpy(hull_side->array+1 , l_max_q, sizeof(Line), hipMemcpyDeviceToDevice));
        hull_side->size = 2;
    }else {
        //points_side->curr_size > 1
        hull_side = combine_hull_par(
                quickhull_split_par(points_side, l_p_max, side),
                quickhull_split_par(points_side, l_max_q, side)
        );
    }

    // free memory
    free_line_par_gpu(l_p_max);
    free_line_par_gpu(l_max_q);
    free_point_array_par_gpu(points_side);

    return hull_side;

}




void workload_calc(size_t* grid_size, size_t* rem_grid_size, size_t* loop_cnt, size_t* sizef, size_t size){

	size_t need_blocks;

	need_blocks = (size + 2*BLOCKSIZE-1)/(2*BLOCKSIZE);
	*loop_cnt = need_blocks/MAX_BLOCK_COUNT;
	
    *sizef = need_blocks*2*BLOCKSIZE;

	if(*loop_cnt < 1){
        *grid_size = need_blocks;
		*rem_grid_size = 0;
		*loop_cnt = 1;	
	}
	else{
        *grid_size = MAX_BLOCK_COUNT; 
		*rem_grid_size = need_blocks%MAX_BLOCK_COUNT;
	}

    

}


Point_array_par* generate_random_points_par(size_t num_of_points, double l_bound, double u_bound){

    time_t t;
    double difference = u_bound - l_bound;
    double offset_x = 0;
    double offset_y = 0;
    Point p;

    srand((unsigned) time(&t));

    Point_array_par* points = init_point_array_par(num_of_points);
    for(size_t i = 0; i < num_of_points; i++){
        offset_x = ((double)rand()/RAND_MAX)*difference;
        offset_y = ((double)rand()/RAND_MAX)*difference;
        p = (Point) {.x = l_bound + offset_x, .y = l_bound + offset_y};
        points->array[i] = p;
    }

    return points;
}


Point_array_par* generate_random_points_on_circle_par(size_t num_of_points, double radius){

    time_t t;
    double angle;
    Point p;

    srand((unsigned) time(&t));

    Point_array_par* points = init_point_array_par(num_of_points);
    for(size_t i = 0; i < num_of_points; i++){      
        angle = ((double)rand() / RAND_MAX) * 2 * M_PI;
        p.x = radius * cos(angle);
        p.y = radius * sin(angle);
        points->array[i] = p;
    }

    printf("Random Point Generation on Circle finished\n");

    return points;
}



Hull_par* combine_hull_par(Hull_par* hull_1, Hull_par* hull_2){

    // vars
    Hull_par* hull_3;
    size_t hull_1_bytes;
    size_t hull_2_bytes;
    size_t hull_3_bytes;

    // set sizes
    hull_1_bytes = hull_1->size*sizeof(Line);
    hull_2_bytes = hull_2->size*sizeof(Line);
    hull_3_bytes = hull_1_bytes+hull_2_bytes;


    // set memory
    hull_3 = init_hull_par_gpu(hull_3_bytes);
    hull_3->size = hull_1->size+hull_2->size;

    // copy results 
    CHECK(hipMemcpy(hull_3->array, hull_1->array, hull_1_bytes, hipMemcpyDeviceToDevice));
    CHECK(hipMemcpy(hull_3->array+hull_1->size, hull_2->array, hull_2_bytes, hipMemcpyDeviceToDevice));

    // free memory
    free_hull_par_gpu(hull_1);
    free_hull_par_gpu(hull_2);

    return hull_3;

}



void points_on_hull_par(Point_array_par* points, Line** l_pq){
    minmax_cuda(points, l_pq);
}






///////////////////////////////////////////////////////////////////////////////
// Stream functions


Hull_par* quickhull_stream_par(Point_array_par* points){

    // device var
    int dev;

    // device set up
    dev = 0;
    CHECK(hipSetDevice(dev));

    Line* l_pq = NULL; //device pointer
    Hull_par* hull_up = NULL;
    Hull_par* hull_down = NULL;
    Hull_par* hull_result_gpu = NULL;
    Hull_par* hull_result_cpu = NULL;

    // above below array var
    Point_array_par* points_gpu;
    Point_array_par* points_above;
    Point_array_par* points_below;

    // stream var
    hipStream_t streams[4];

    // set memory
    points_gpu = init_point_array_par_gpu(points->size);
    points_above = init_point_array_par_gpu(0);
    points_below = init_point_array_par_gpu(0);

    // transfer points
    #if MEMORY_MODEL == ZERO_MEMORY
        points_gpu = init_point_array_par_gpu(0);
        points_gpu->size = points->size;
        CHECK(hipHostGetDevicePointer((void **)&points_gpu->array, (void *)points->array, 0));
    #else
        points_gpu = init_point_array_par_gpu(points->size);
        CHECK(hipMemcpy(points_gpu->array, points->array, points->size*sizeof(Point), hipMemcpyHostToDevice));
    #endif


    // init streams
    for(int i = 0; i < 4; i++){
         hipStreamCreate(&streams[i]);
    }



    // find points on hull
    points_on_hull_stream_par(points_gpu, &l_pq, streams);
    //l_pq = (Line) { .p = p, .q = q };

    // splits array into above and below
    split_stream_point_array(points_gpu, points_above, points_below, l_pq, streams);

    // free memory
    #if MEMORY_MODEL != ZERO_MEMORY
        free_point_array_stream_par_gpu(points_gpu, streams[1]);
    #endif
    

    // recursive call
    #pragma omp parallel num_threads(2)
    {
        #pragma omp single
        {
            #pragma omp task
            {
                hull_up = first_quickhull_stream_split_par(points_above, l_pq, ABOVE, &streams[0]);
            }

            #pragma omp task
            {
                hull_down = first_quickhull_stream_split_par(points_below, l_pq, BELOW, &streams[2]);
            }
        }
    }

    // Sync
    hipDeviceSynchronize();

    // combine
    hull_result_gpu = combine_hull_stream_par(hull_up, hull_down, streams);

    // copy back results
    hull_result_cpu = init_hull_par(hull_result_gpu->size);
    CHECK(hipMemcpy(hull_result_cpu->array, hull_result_gpu->array, hull_result_gpu->size*sizeof(Line), hipMemcpyDeviceToHost));


    // destroy streams
    for(int i = 0; i < 4; i++){
         hipStreamDestroy(streams[i]);
    }
    
    // free memory
    free_line_par_gpu(l_pq);
    free_hull_par_gpu(hull_result_gpu);
    free_point_array_par_gpu(points_above);
    free_point_array_par_gpu(points_below);

    return hull_result_cpu;

}

Hull_par* first_quickhull_stream_split_par(Point_array_par* points, Line* l, int side, hipStream_t* streams){

    // vars
    Point_array_par* points_side = NULL;
    Line* l_p_max = NULL;
    Line* l_max_q = NULL;
    Hull_par* hull_side = NULL;

    // set memory
    points_side = points;

    // find point with max distance
    max_distance_stream_cuda(l, points_side, &l_p_max, &l_max_q, streams); // returns l_p_max and l_max_q gpu mem pointer, l is a device pointer
    // l_p_max = (Line) { .p = l.p, .q = max_point };
    // l_max_q = (Line) { .p = max_point, .q = l.q };


    if(points_side->size == 0) {
        hull_side = init_hull_par_gpu(1);
        CHECK(hipMemcpyAsync(hull_side->array , l, sizeof(Line), hipMemcpyDeviceToDevice, streams[0]));
        hull_side->size = 1;
    }else if(points_side->size == 1){
        hull_side = init_hull_par_gpu(2);
        CHECK(hipMemcpyAsync(hull_side->array , l_p_max, sizeof(Line), hipMemcpyDeviceToDevice, streams[0]));
        CHECK(hipMemcpyAsync(hull_side->array+1 , l_max_q, sizeof(Line), hipMemcpyDeviceToDevice, streams[1]));
        hull_side->size = 2;

    }else {
        //points_side->curr_size > 1
        hull_side = combine_hull_stream_par(
                quickhull_stream_split_par(points_side, l_p_max, side, streams),
                quickhull_stream_split_par(points_side, l_max_q, side, streams),
                streams
        );
    }

    // free memory
    free_line_stream_par_gpu(l_p_max, streams[0]);
    free_line_stream_par_gpu(l_max_q, streams[1]);

    return hull_side;

}


Hull_par* quickhull_stream_split_par(Point_array_par* points, Line* l, int side, hipStream_t* streams){

    // vars
    Point_array_par* points_side = NULL;
    Line* l_p_max = NULL;
    Line* l_max_q = NULL;
    Hull_par* hull_side = NULL;

    // set memory
    points_side = init_point_array_par_gpu(0);

    // split array
    split_stream_point_array_side(points, points_side, l, side, streams);

    // find point with max distance
    max_distance_stream_cuda(l, points_side, &l_p_max, &l_max_q, streams); // returns l_p_max and l_max_q gpu mem pointer
    // l_p_max = (Line) { .p = l.p, .q = max_point };
    // l_max_q = (Line) { .p = max_point, .q = l.q };

    if(points_side->size == 0) {
        hull_side = init_hull_par_gpu(1);
        CHECK(hipMemcpyAsync(hull_side->array , l, sizeof(Line), hipMemcpyDeviceToDevice, streams[0]));
        hull_side->size = 1;
    }else if(points_side->size == 1){
        hull_side = init_hull_par_gpu(2);
        CHECK(hipMemcpyAsync(hull_side->array , l_p_max, sizeof(Line), hipMemcpyDeviceToDevice, streams[0]));
        CHECK(hipMemcpyAsync(hull_side->array+1 , l_max_q, sizeof(Line), hipMemcpyDeviceToDevice, streams[1]));
        hull_side->size = 2;
    }else {
        //points_side->curr_size > 1
        hull_side = combine_hull_stream_par(
                quickhull_stream_split_par(points_side, l_p_max, side, streams),
                quickhull_stream_split_par(points_side, l_max_q, side, streams),
                streams
        );
    }

    // free memory
    free_line_stream_par_gpu(l_p_max, streams[0]);
    free_line_stream_par_gpu(l_max_q, streams[1]);
    free_point_array_stream_par_gpu(points_side, streams[0]);

    return hull_side;

}



void points_on_hull_stream_par(Point_array_par* points, Line** l_pq, hipStream_t* streams){
    minmax_stream_cuda(points, l_pq, streams);
}




Hull_par* combine_hull_stream_par(Hull_par* hull_1, Hull_par* hull_2, hipStream_t* streams){

    // vars
    Hull_par* hull_3;
    size_t hull_1_bytes;
    size_t hull_2_bytes;
    size_t hull_3_bytes;

    // set sizes
    hull_1_bytes = hull_1->size*sizeof(Line);
    hull_2_bytes = hull_2->size*sizeof(Line);
    hull_3_bytes = hull_1_bytes+hull_2_bytes;


    // set memory
    hull_3 = init_hull_par_gpu(hull_3_bytes);
    hull_3->size = hull_1->size+hull_2->size;

    // copy results 
    CHECK(hipMemcpyAsync(hull_3->array, hull_1->array, hull_1_bytes, hipMemcpyDeviceToDevice, streams[0]));
    CHECK(hipMemcpyAsync(hull_3->array+hull_1->size, hull_2->array, hull_2_bytes, hipMemcpyDeviceToDevice, streams[1]));

    // free memory
    free_hull_stream_par_gpu(hull_1, streams[0]);
    free_hull_stream_par_gpu(hull_2, streams[1]);

    return hull_3;

}
